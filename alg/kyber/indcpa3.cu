#include "hip/hip_runtime.h"

// @Author: Arpan Jati
// Adapted from NewHope Reference Codebase and Parallelized using CUDA
// Updated : August 2019

#include <stdint.h>
#include <malloc.h>
#include "indcpa3.h"
#include "poly3.h"
#include "polyvec3.h"
#include "rng3.h"
#include "ntt3.h"
#include "symmetric3.h"

#include <stdlib.h>
#include <float.h>




void print_data(const char* text, unsigned char* data, int length)
{
	printf("%s\n", text);

	for (int i = 0; i < length; i++)
	{
		printf("%02X", data[i]);

		if ((i + 1) % 2 == 0)
		{
			printf(" ");
		}

		if ((i + 1) % 32 == 0)
		{
			printf("\n");
		}
	}

	printf("\n");
}

__device__ void print_data_d(const char* text, unsigned char* data, int length)
{
	printf("%s\n", text);

	for (int i = 0; i < length; i++)
	{
		printf("%02X", data[i]);

		if ((i + 1) % 2 == 0)
		{
			printf(" ");
		}

		if ((i + 1) % 32 == 0)
		{
			printf("\n");
		}
	}

	printf("\n");

}



/*************************************************
* Name:        pack_pk
*
* Description: Serialize the public key as concatenation of the
*              serialized vector of polynomials pk
*              and the public seed used to generate the matrix A.
*
* Arguments:   unsigned char *r:          pointer to the output serialized public key
*               poly *pk:            pointer to the input public-key polynomial
*               unsigned char *seed: pointer to the input public seed
**************************************************/
__global__  void pack_pk_n(int COUNT, unsigned char* r, polyvec* pk, unsigned char* seed)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_seed = KYBER_SYMBYTES * X;
		int o_r = KYBER_INDCPA_PUBLICKEYBYTES * X;

		int i;
		polyvec_tobytes(r + o_r, pk);
		for (i = 0; i < KYBER_SYMBYTES; i++)
			(r + o_r)[i + KYBER_POLYVECBYTES] = (seed + o_seed)[i];
	}
}

/*************************************************
* Name:        unpack_pk
*
* Description: De-serialize public key from a byte array;
*              approximate inverse of pack_pk
*
* Arguments:   - polyvec *pk:                   pointer to output public-key vector of polynomials
*              - unsigned char *seed:           pointer to output seed to generate matrix A
*              -  unsigned char *packedpk: pointer to input serialized public key
**************************************************/
__global__  void unpack_pk_n(int COUNT, polyvec* pk, unsigned char* seed, unsigned char* packedpk)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_packedpk = KYBER_INDCPA_PUBLICKEYBYTES * X;
		int o_seed = KYBER_SYMBYTES * X;

		int i;
		polyvec_frombytes(pk, packedpk + o_packedpk);
		for (i = 0; i < KYBER_SYMBYTES; i++)
			(seed + o_seed)[i] = (packedpk + o_packedpk)[i + KYBER_POLYVECBYTES];
	}
}

/*************************************************
* Name:        pack_sk
*
* Description: Serialize the secret key
*
* Arguments:   - unsigned char *r:  pointer to output serialized secret key
*              -  polyvec *sk: pointer to input vector of polynomials (secret key)
**************************************************/
__global__  void pack_sk_n(int COUNT, unsigned char* r, polyvec* sk)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_r = KYBER_INDCPA_SECRETKEYBYTES * X;

		polyvec_tobytes(r + o_r, sk);
	}
}

/*************************************************
* Name:        unpack_sk
*
* Description: De-serialize the secret key;
*              inverse of pack_sk
*
* Arguments:   - polyvec *sk:                   pointer to output vector of polynomials (secret key)
*              -  unsigned char *packedsk: pointer to input serialized secret key
**************************************************/
__global__  void unpack_sk_n(int COUNT, polyvec* sk, unsigned char* packedsk)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_r = KYBER_INDCPA_SECRETKEYBYTES * X;

		polyvec_frombytes(sk, packedsk + o_r);
	}
}

/*************************************************
* Name:        pack_ciphertext
*
* Description: Serialize the ciphertext as concatenation of the
*              compressed and serialized vector of polynomials b
*              and the compressed and serialized polynomial v
*
* Arguments:   unsigned char *r:          pointer to the output serialized ciphertext
*               poly *pk:            pointer to the input vector of polynomials b
*               unsigned char *seed: pointer to the input polynomial v
**************************************************/
__global__  void pack_ciphertext_n(int COUNT, unsigned char* r, polyvec* b, poly3* v)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_r = X * KYBER_INDCPA_BYTES;

		polyvec_compress((r + o_r), b);
		poly_compress((r + o_r) + KYBER_POLYVECCOMPRESSEDBYTES, v);
	}
}

/*************************************************
* Name:        unpack_ciphertext
*
* Description: De-serialize and decompress ciphertext from a byte array;
*              approximate inverse of pack_ciphertext
*
* Arguments:   - polyvec *b:             pointer to the output vector of polynomials b
*              - poly *v:                pointer to the output polynomial v
*              -  unsigned char *c: pointer to the input serialized ciphertext
**************************************************/
__global__  void unpack_ciphertext_n(int COUNT, polyvec* b, poly3* v, unsigned char* c)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		int o_c = X * KYBER_INDCPA_BYTES;

		polyvec_decompress(b, (c + o_c));
		poly_decompress(v, (c + o_c) + KYBER_POLYVECCOMPRESSEDBYTES);
	}
}

/*************************************************
* Name:        rej_uniform
*
* Description: Run rejection sampling on uniform random bytes to generate
*              uniform random integers mod q
*
* Arguments:   - int16_t *r:               pointer to output buffer
*              - unsigned int len:         requested number of 16-bit integers (uniform mod q)
*              - unsigned char *buf: pointer to input buffer (assumed to be uniform random bytes)
*              - unsigned int buflen:      length of input buffer in bytes
*
* Returns number of sampled 16-bit integers (at most len)
**************************************************/
__device__  unsigned int rej_uniform(poly3* r, unsigned int len,
	unsigned char* buf, unsigned int buflen)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;

	unsigned int ctr, pos;
	uint16_t val;

	//printf("\n UNIFORM G:\n ");

	ctr = pos = 0;
	while (ctr < len && pos + 2 <= buflen)
	{
		val = buf[pos] | ((uint16_t)buf[pos + 1] << 8);
		pos += 2;

		if (val < 19 * KYBER_Q)
		{
			val -= (val >> 12) * KYBER_Q; // Barrett reduction
			r->coeffs[ctr++].threads[X] = (int16_t)val;

			//printf(" %5d = %5d | ", (ctr - 1), val);

			//if ((((ctr - 1) + 1) % 8) == 0)
			//{
				//printf("\n");
			//}

		}
	}

	return ctr;
}

//#define gen_a(A,B)  gen_matrix(A,B,0)
//#define gen_at(A,B) gen_matrix(A,B,1)

/*************************************************
* Name:        gen_matrix
*
* Description: Deterministically generate matrix A (or the transpose of A)
*              from a seed. Entries of the matrix are polynomials that look
*              uniformly random. Performs rejection sampling on output of
*              a XOF
*
* Arguments:   - polyvec *a:                pointer to ouptput matrix A
*              -  unsigned char *seed: pointer to input seed
*              - int transposed:            boolean deciding whether A or A^T is generated
**************************************************/
__global__ void gen_matrix_n(int COUNT, polyvec* a,
	unsigned char* seed, int transposed, unsigned char* large_bufA) // Not static for benchmarking
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		unsigned int ctr, i, j;
		unsigned int maxnblocks = (530 + XOF_BLOCKBYTES) / XOF_BLOCKBYTES; /* 530 is expected number of required bytes */

	   // int buf_bytes = XOF_BLOCKBYTES * maxnblocks + 1;

		int o_largeBuffer = X * LARGE_BUFFER_SZ;


		unsigned char* buf = (large_bufA + o_largeBuffer);//(unsigned char*)malloc(buf_bytes);

		xof_state state;

		//printf("\n gen_matrix START ---------------------\n ");

		int o_seed = KYBER_SYMBYTES * X;

		for (i = 0; i < KYBER_K; i++)
		{
			for (j = 0; j < KYBER_K; j++)
			{
				if (transposed)
				{
					kyber_shake128_absorb(&state, seed + o_seed, i, j);
				}
				else
				{
					kyber_shake128_absorb(&state, seed + o_seed, j, i);
				}

				kyber_shake128_squeezeblocks(buf, maxnblocks, &state);

				ctr = rej_uniform(&(a[i].vec[j]), KYBER_N, buf, maxnblocks * XOF_BLOCKBYTES);

				//printf("\n I:%d | J: %d | CTR: %d", i, j, ctr);

				while (ctr < KYBER_N)
				{
					kyber_shake128_squeezeblocks(buf, 1, &state);

					ctr += rej_uniform(&(a[i].vec[j]) + (ctr * N_TESTS),
						KYBER_N - ctr, buf, XOF_BLOCKBYTES);

					//printf("\n I:%d | J: %d | CTR: %d", i, j, ctr);
				}

				// print_data_d("\n\n\n buf", buf, XOF_BLOCKBYTES * maxnblocks + 1);

				//__syncthreads();

			}
		}

		//free(buf);

		//printf("\n gen_matrix END ---------------------\n ");

	}

}

int blockSize = BLOCK_SIZE;
int gridSize = (N_TESTS + blockSize - 1) / blockSize;

//int blockSize = N_TESTS;
//int gridSize = 1;

__global__ void print_poly(int COUNT, poly3* poly)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		printf("GPU | PRINT POLY\n");

		for (int i = 0; i < 256; i++)
		{
			printf("%d ", poly->coeffs[i].threads[0]);

			//if ((i + 1) % 2 == 0)
			//{
			//	printf(" ");
			//}

			if ((i + 1) % 8 == 0)
			{
				printf("\n");
			}
		}

		printf("\n");

	}
}


__global__ void print_polyvec(int COUNT, polyvec* polyvec)
{
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	if (X < COUNT)
	{
		printf("GPU POLYVEC ---------------------------\n");

		for (int v = 0; v < KYBER_K; v++)
		{
			printf("VEC---- %d\n", v);

			for (int i = 0; i < 256; i++)
			{
				printf("%d ", (polyvec->vec[v]).coeffs[i].threads[X]);

				if ((i + 1) % 8 == 0)
				{
					printf("\n");
				}
			}
		}

		printf("\n");
	}
}

/*************************************************
* Name:        indcpa_keypair
*
* Description: Generates public and private key for the CPA-secure
*              public-key encryption scheme underlying Kyber
*
* Arguments:   - unsigned char *pk: pointer to output public key (of length KYBER_INDCPA_PUBLICKEYBYTES bytes)
*              - unsigned char *sk: pointer to output private key (of length KYBER_INDCPA_SECRETKEYBYTES bytes)
**************************************************/
void indcpa_keypair(int COUNT, poly_set4* ps, unsigned char* pk, unsigned char* sk,
	unsigned char* rng_buf, hipStream_t stream)
{
	//printf("\nCOUNT: %d \n", COUNT);
	//printf("DEFAULT BLOCK SIZE: %d \n", blockSize);

	polyvec* a = ps->AV; // [KYBER_K]

	polyvec* e = ps->av;
	polyvec* pkpv = ps->bv;
	polyvec* skpv = ps->cv;

	poly3* poly_temp = ps->a;

	// unsigned char* buf = ps->seed;

	unsigned char* publicseed = rng_buf;
	unsigned char* noiseseed = rng_buf + KYBER_SYMBYTES;
	int i;
	unsigned char nonce = 0;

	unsigned char* large_bufA = ps->large_buffer_a;

	//randombytes_device(buf, KYBER_SYMBYTES);

		sha3_512_n << < gridSize, blockSize, 0, stream >> > (COUNT, rng_buf, rng_buf, KYBER_SYMBYTES);



		gen_matrix_n << <gridSize, blockSize, 0, stream >> > (COUNT, a, publicseed, 0, large_bufA);


		for (i = 0; i < KYBER_K; i++)
		{
	
				poly_getnoise << <gridSize, blockSize, 0, stream >> > (COUNT, skpv->vec + i, noiseseed, nonce++);
	
		}

	for (i = 0; i < KYBER_K; i++)
	{

			poly_getnoise << <gridSize, blockSize, 0, stream >> > (COUNT, e->vec + i, noiseseed, nonce++);

	}

	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(polyvec_ntt_n), hipFuncCachePreferL1);

		polyvec_ntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, skpv);

		polyvec_ntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, e);
	

		//print_polyvec << <gridSize, blockSize >> > (skpv);
		//print_polyvec << <gridSize, blockSize >> > (e);

		// matrix-vector multiplication
		for (i = 0; i < KYBER_K; i++)
		{
	
				polyvec_pointwise_acc_n << <gridSize, blockSize, 0, stream >> > (COUNT, &(pkpv->vec[i]), &(a[i]), skpv, poly_temp);
	
				poly_frommont_n << <gridSize, blockSize, 0, stream >> > (COUNT, &(pkpv->vec[i]));

				//print_poly << <gridSize, blockSize,  0, stream>> > (&(pkpv->vec[i]));
		}

	//print_polyvec << <gridSize, blockSize,  0, stream>> > (pkpv);
	//print_polyvec << <gridSize, blockSize,  0, stream>> > (e);


		polyvec_add_n << <gridSize, blockSize, 0, stream >> > (COUNT, pkpv, pkpv, e);

		polyvec_reduce_n << <gridSize, blockSize, 0, stream >> > (COUNT, pkpv);

		pack_sk_n << <gridSize, blockSize, 0, stream >> > (COUNT, sk, skpv);

		pack_pk_n << <gridSize, blockSize, 0, stream >> > (COUNT, pk, pkpv, publicseed);

}

/*************************************************
* Name:        indcpa_enc
*
* Description: Encryption function of the CPA-secure
*              public-key encryption scheme underlying Kyber.
*
* Arguments:   - unsigned char *c:          pointer to output ciphertext (of length KYBER_INDCPA_BYTES bytes)
*              -  unsigned char *m:    pointer to input message (of length KYBER_INDCPA_MSGBYTES bytes)
*              -  unsigned char *pk:   pointer to input public key (of length KYBER_INDCPA_PUBLICKEYBYTES bytes)
*              -  unsigned char *coin: pointer to input random coins used as seed (of length KYBER_SYMBYTES bytes)
*                                           to deterministically generate all randomness
**************************************************/
void indcpa_enc(int COUNT, poly_set4* ps, unsigned char* c,
	unsigned char* m,
	unsigned char* pk,
	unsigned char* coins, hipStream_t stream)
{
	polyvec* at = ps->AV; // [KYBER_K]

	polyvec* sp = ps->av;
	polyvec* pkpv = ps->bv;
	polyvec* ep = ps->cv;
	polyvec* bp = ps->dv;

	poly3* v = ps->a;
	poly3* k = ps->b;
	poly3* epp = ps->c;

	poly3* poly_temp = ps->d;

	unsigned char* seed = ps->seed;
	int i;
	unsigned char nonce = 0;

	unsigned char* large_bufA = ps->large_buffer_a;


		unpack_pk_n << <gridSize, blockSize, 0, stream >> > (COUNT, pkpv, seed, pk);

		poly_frommsg_n << <gridSize, blockSize, 0, stream >> > (COUNT, k, m);

		gen_matrix_n << <gridSize, blockSize, 0, stream >> > (COUNT, at, seed, 1, large_bufA);


		for (i = 0; i < KYBER_K; i++)
		{
		
				poly_getnoise << <gridSize, blockSize,  0, stream >> > (COUNT, sp->vec + i, coins, nonce++);
		
		}

	for (i = 0; i < KYBER_K; i++)
	{
	
			poly_getnoise << <gridSize, blockSize, 0, stream >> > (COUNT, ep->vec + i, coins, nonce++);
		
	}

		poly_getnoise << <gridSize, blockSize, 0, stream >> > (COUNT, epp, coins, nonce++);

		polyvec_ntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, sp);

		// matrix-vector multiplication
		for (i = 0; i < KYBER_K; i++)
		{

				polyvec_pointwise_acc_n << <gridSize, blockSize, 0, stream >> > (COUNT, &(bp->vec[i]), &(at[i]), sp, poly_temp);

		}


		polyvec_pointwise_acc_n << <gridSize, blockSize, 0, stream >> > (COUNT, v, pkpv, sp, poly_temp);

		polyvec_invntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, bp);

		poly_invntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, v);

		polyvec_add_n << <gridSize, blockSize, 0, stream >> > (COUNT, bp, bp, ep);

		poly_add_n << <gridSize, blockSize, 0, stream >> > (COUNT, v, v, epp);

		poly_add_n << <gridSize, blockSize, 0, stream >> > (COUNT, v, v, k);

		polyvec_reduce_n << <gridSize, blockSize, 0, stream >> > (COUNT, bp);

		poly_reduce_n << <gridSize, blockSize, 0, stream >> > (COUNT, v);

		pack_ciphertext_n << <gridSize, blockSize, 0, stream >> > (COUNT, c, bp, v);

}

/*************************************************
* Name:        indcpa_dec
*
* Description: Decryption function of the CPA-secure
*              public-key encryption scheme underlying Kyber.
*
* Arguments:   - unsigned char *m:        pointer to output decrypted message (of length KYBER_INDCPA_MSGBYTES)
*              -  unsigned char *c:  pointer to input ciphertext (of length KYBER_INDCPA_BYTES)
*              -  unsigned char *sk: pointer to input secret key (of length KYBER_INDCPA_SECRETKEYBYTES)
**************************************************/
void indcpa_dec(int COUNT, poly_set4* ps, unsigned char* m,
	unsigned char* c,
	unsigned char* sk, hipStream_t stream)
{
	polyvec* bp = ps->av;
	polyvec* skpv = ps->bv;

	poly3* v = ps->a;
	poly3* mp = ps->b;
	poly3* poly_temp = ps->c;


		unpack_ciphertext_n << <gridSize, blockSize, 0, stream >> > (COUNT, bp, v, c);

		unpack_sk_n << <gridSize, blockSize, 0, stream >> > (COUNT, skpv, sk);

		polyvec_ntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, bp);

		polyvec_pointwise_acc_n << <gridSize, blockSize, 0, stream >> > (COUNT, mp, skpv, bp, poly_temp);

		poly_invntt_n << <gridSize, blockSize, 0, stream >> > (COUNT, mp);

		poly_sub_n << <gridSize, blockSize, 0, stream >> > (COUNT, mp, v, mp);

		poly_reduce_n << <gridSize, blockSize, 0, stream >> > (COUNT, mp);

		poly_tomsg_n << <gridSize, blockSize, 0, stream >> > (COUNT, m, mp);

}
